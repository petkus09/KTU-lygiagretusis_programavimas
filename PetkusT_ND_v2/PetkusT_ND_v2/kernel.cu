#include "hip/hip_runtime.h"
//IFF-1 Tautvydas Petkus
//L-ND - CUDA
//Failo dydis - 100x100 duomen�
//Dabartiniai nustatymai: stulpeli� skai�ius - 100, eilu�i� skai�ius - 100

#include "hip/hip_runtime.h"
#include ""

#include <time.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>	
#include <fstream>	
#include <sstream>	
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <time.h>

using namespace std;
const int MAX_ROW = 108;	   //MAX 108
const int MAX_COL = 330;	   //MAX 330
const int counter_hit = 100;

struct bool_data{
	bool data[MAX_COL];
};

void evolution(bool (*pop)[MAX_COL], int generation);
void print(bool (*pop)[MAX_COL], int generation);
bool generation_pass(bool (*pop)[MAX_COL], int i, int j);
bool generation_pass_paralell(bool_data *pop, int i, int j);
int check_neighbour(bool pop);

hipError_t cudaEvolution(bool (*pop)[MAX_COL], int generation, char mode);

__device__ int check_neighbour_paralell(bool pop)
{
	if (pop)
	{
		return 1;
	}
	else
	{
		return 0;
	}
}

__device__ bool generation_pass_paralell(bool_data *pop, int i, int j)
{
	int counter = 0;
	if (i != 0) {counter += check_neighbour_paralell(pop[i-1].data[j]); }
	if (i != 0 && j != MAX_COL - 1) {counter += check_neighbour_paralell(pop[i-1].data[j + 1]); }
	if (j != MAX_COL - 1) {counter += check_neighbour_paralell(pop[i].data[j + 1]); }
	if (i != MAX_ROW - 1 && j != MAX_COL - 1) {counter += check_neighbour_paralell(pop[i + 1].data[j + 1]); }
	if (i != MAX_ROW - 1) {counter += check_neighbour_paralell(pop[i + 1].data[j]); }
	if (i != MAX_ROW - 1 && j != 0) {counter += check_neighbour_paralell(pop[i+1].data[j-1]); }
	if (j != 0) {counter += check_neighbour_paralell(pop[i].data[j-1]); }
	if (i != 0 && j != 0) {counter += check_neighbour_paralell(pop[i-1].data[j-1]); }
	if (pop[i].data[j])
	{
		if (counter < 2)
		{
			return false;
		}
		else if (counter > 3)
		{
			return false;
		}
		else
		{
			return true;
		}
	}
	else
	{
		if (counter == 3)
		{
			return true;
		}
	}
	return false;
}

__device__ void print_paralell(bool_data *pop)
{
	//system("cls");
	char row[MAX_ROW*MAX_COL+MAX_ROW];
	int offset = 0;
	for (int i = 0; i < MAX_ROW; i++)
	{
		for (int j = 0; j < MAX_COL; j++)
		{
			if (pop[i].data[j])
			{
				row[offset] = '0';
			}
			else{
				row[offset] = ' ';
			}
			offset += 1;
		}
		row[offset] = '\n';
		offset += 1;
	}
	//printf("%s\n", row);
}

__global__ void startEvolution(bool_data *p, bool_data *new_p){                 //1 iteracija
	int row = blockIdx.x;
	int col = threadIdx.x;
	new_p[row].data[col] = generation_pass_paralell(p, row, col);
}

__global__ void startEvolutionCHAOS(bool_data *p, bool_data *new_p){              //100 iteraciju
	int row = blockIdx.x;
	int col = threadIdx.x;
	for (int i = 0; i < 100; i++)
	{
		new_p[row].data[col] = generation_pass_paralell(p, row, col);
	}
}

int main()
{
	char ch;
	bool population[MAX_ROW][MAX_COL];
	printf("Press 1 for random generator, press 2 for file input...\n");
	ch = fgetchar();
	if (ch == '1')
	{
		srand(time(NULL));
		for (int i = 0; i < MAX_ROW; i++)
		{
			for (int j = 0; j < MAX_COL; j++)
			{
				if (rand() % 2)
				{
					population[i][j] = false;
				}
				else
				{
					population[i][j] = true;
				}
			}
		}
	}
	else if (ch == '2')
	{
		ifstream in("PetkusT.txt");
		for(string line; getline(in, line);){	
			for (int i = 0; i < MAX_ROW; i++)
			{
				getline(in, line);
				if (line != ""){
					stringstream ss;
					ss << line;
					int k = 0;
					for (int j = 0; j < MAX_COL; j++)
					{
						ss >> k;
						if (k == 1){
							population[i][j] = true;
						}
						else{
							population[i][j] = false;
						}
					}
				}
			}
		}
	}
	int generation = 0;
	print(population, generation);
	printf("Press 1 for sequental game, press 2 parallel game, press 3 for CHAOS mode...\n");
	ch = fgetchar(); ch = fgetchar();
	printf("Press any key to continue...");
	getchar(); getchar();
	int counter = 0;
	clock_t begin, end;
	double time_spent;
	begin = clock();
	if (ch == '1')
	{
		//while (counter < counter_hit)
		while (true)
		{
			evolution(population, generation);	//Nuosekliai
			generation += 1;
			counter += 1;
		}
	}
	else if (ch == '2' || ch == '3')
	{
		//while (counter < counter_hit)
		while (true)
		{
			hipError_t cudaStatus = cudaEvolution(population, generation, ch);	//Ivykdome funkcija, kurioje algoritmas bus atliekamas lygiagreciai
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "cudaEvolution failed!");
				fgetchar();
				return 1;
			}
			cudaStatus = hipDeviceReset();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipDeviceReset failed!");
				return 1;
			}
			generation += 1;
			if (ch == '2')
			{
				counter += 1;
			}
			else if (ch == '3')
			{
				counter += 100;
			}
		}
	}
	end = clock();
	time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
	printf("\n%f\n", time_spent);
	getchar();
	return 0;
}

void evolution(bool (*pop)[MAX_COL], int generation)
{
	bool new_pop[MAX_ROW][MAX_COL];
	for (int i = 0; i < MAX_ROW; i++)
	{
		for (int j = 0; j < MAX_COL; j++)
		{
			new_pop[i][j] = generation_pass(pop, i, j);
		}
	}
	memcpy(pop,new_pop, MAX_ROW*MAX_COL*sizeof(bool));
	print(pop, generation);
}

void print(bool (*pop)[MAX_COL], int generation)
{
	system("cls");
	char row[MAX_ROW*MAX_COL+MAX_ROW];
	printf("Generation: %5d\n", generation);
	int offset = 0;
	for (int i = 0; i < MAX_ROW; i++)
	{
		for (int j = 0; j < MAX_COL; j++)
		{
			if (pop[i][j])
			{
				row[offset] = '0';
			}
			else{
				row[offset] = ' ';
			}
			offset += 1;
		}
		row[offset] = '\n';
		offset += 1;
	}
	printf("%s\n", row);
}

bool generation_pass(bool (*pop)[MAX_COL], int i, int j)
{
	int counter = 0;
	if (i != 0) {counter += check_neighbour(pop[i-1][j]); }
	if (i != 0 && j != MAX_COL - 1) {counter += check_neighbour(pop[i-1][j + 1]); }
	if (j != MAX_COL - 1) {counter += check_neighbour(pop[i][j + 1]); }
	if (i != MAX_ROW - 1 && j != MAX_COL - 1) {counter += check_neighbour(pop[i + 1][j + 1]); }
	if (i != MAX_ROW - 1) {counter += check_neighbour(pop[i + 1][j]); }
	if (i != MAX_ROW - 1 && j != 0) {counter += check_neighbour(pop[i+1][j-1]); }
	if (j != 0) {counter += check_neighbour(pop[i][j-1]); }
	if (i != 0 && j != 0) {counter += check_neighbour(pop[i-1][j-1]); }
	if (pop[i][j])
	{
		if (counter < 2)
		{
			return false;
		}
		else if (counter > 3)
		{
			return false;
		}
		else
		{
			return true;
		}
	}
	else
	{
		if (counter == 3)
		{
			return true;
		}
	}
	return false;
}

int check_neighbour(bool pop)
{
	if (pop)
	{
		return 1;
	}
	else
	{
		return 0;
	}
}

hipError_t cudaEvolution(bool (*pop)[MAX_COL], int generation, char mode)
{
	struct bool_data p[MAX_ROW];
	struct bool_data new_p[MAX_ROW];
	for (int i = 0; i < MAX_ROW; i++)
	{
		for (int j = 0; j < MAX_COL; j++)
		{
			p[i].data[j] = pop[i][j];
		}
	}
	struct bool_data *dev_p;
	struct bool_data *dev_new_p;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_p, MAX_ROW *  sizeof(bool_data));	//skiriame atminti tiek pradinei matricai
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_new_p, MAX_ROW *  sizeof(bool_data));	//tiek naujai matricai
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_p, p, MAX_ROW *  sizeof(bool_data), hipMemcpyHostToDevice);	//perduodam duomenis	
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_new_p, new_p, MAX_ROW *  sizeof(bool_data), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	if (mode == '2')
	{
		startEvolution<<<MAX_ROW, MAX_COL>>>(dev_p, dev_new_p);		//Ivykdom gijas
	}
	if (mode == '3')
	{
		startEvolutionCHAOS<<<MAX_ROW + 1, MAX_COL>>>(dev_p, dev_new_p);
	}


	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "startEvolution launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMemcpy(p, dev_p,   MAX_ROW *  sizeof(bool_data), hipMemcpyDeviceToHost);		//Susigrazinam rezultatus
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(new_p, dev_new_p,   MAX_ROW *  sizeof(bool_data), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	for (int i = 0; i < MAX_ROW; i++)
	{
		for (int j = 0; j < MAX_COL; j++)
		{
			pop[i][j] = new_p[i].data[j];		//perrasom is naujo pradine matrica
		}
	}
	print(pop, generation);
Error:
	hipFree(dev_p);
	return cudaStatus;
}
