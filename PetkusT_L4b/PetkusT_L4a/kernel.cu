//IFF-1 Tautvydas Petkus
// L4b - Thrust
//Failo dydis - 50 eiluciu
//Dabartiniai nustatymai: giju sk: 8, maximalus masyvo dydis - 10, didziausias char buferio dydis - 80
//Kiek  iteracij� i� eil�s padaro vienas procesas? viena pilnai
//Kokia tvarka vykdomi procesai? tokia, kokia startuoja
//

#include "hip/hip_runtime.h"
#include ""

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>	
#include <fstream>	
#include <sstream>	
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/sequence.h> 

using namespace std;

const int MAX_STRING_LEN = 70;
const int MAX_THREADS = 5;
const int MAX_FILE_ROW = 50;
const int MAX_ARRAY_SIZE = 5;

struct Data
{
	char text_var[MAX_STRING_LEN];
	int int_var;
	double double_var;
};

struct ThreadData{
	struct Data thread_struct_array[MAX_ARRAY_SIZE];
	int n;
};

Data Plius(Data D1, Data D2, Data D3, Data D4, Data D5);
void addWithCuda(Data *c, ThreadData *a);

int main()
{
	//pradiniu duomenu kintamieji
	int array_size[MAX_THREADS] = {5, 5, 5, 5, 5};
	ifstream in("PetkusT.txt");
	struct Data duomenys[MAX_FILE_ROW];
	struct ThreadData duomenys_gijoms[MAX_THREADS];
	//Skaitomas duomenu failas
	for(string line; getline(in, line);){	
		for (int i = 0; i < MAX_FILE_ROW; i++)
		{
			getline(in, line);
			if (line != ""){
				stringstream ss;
				ss << line;
				Data eilute;
				char string_var[16];
				ss >> string_var >> eilute.int_var >> eilute.double_var;

				strcpy(eilute.text_var, string_var);
				duomenys[i] = eilute;
			}
		}
	}
	//pradiniu duomenu atspausdinimas
	int d = 0;
	int ii = 0;
	int j = 0;
	printf("********************************************************************\n");
	printf("***Pradiniai duomenys***\n");
	for (ii = 0; ii < MAX_THREADS; ii++){
		printf("***Gija nr. %d***\n", ii + 1);
		printf("%10s %10s %10s %10s\n", "Eil.Nr.", "String", "int", "double");
		struct Data D_gija[MAX_ARRAY_SIZE];
		for (j = 0; j < array_size[ii]; j++){
			D_gija[j] = duomenys[d];
			printf("%10d %10s %10d %10lf\n", j + 1, D_gija[j].text_var, D_gija[j].int_var, D_gija[j].double_var);
			d++;

		}
		for (j = 0; j < array_size[ii]; j++){
			duomenys_gijoms[ii].thread_struct_array[j] = D_gija[j];
			duomenys_gijoms[ii].n = array_size[ii];
		}
	}
	printf("\n**********************\n");
	//Lygiagrecioji dalis

	printf("***Lygiagrecioji programos dalis***\n");
	printf("%10s %10s %10s\n", "String", "int", "double");
	struct Data result_array[MAX_ARRAY_SIZE];
	for (int i = 0; i < MAX_ARRAY_SIZE; i++)
	{
		Data duom;
		strcpy(duom.text_var, "");
		duom.int_var = 0;
		duom.double_var = 0.0;
		result_array[i] = duom;
	}
	for (int i = 0; i < MAX_THREADS; i++)
	{
		for (int j = 0; j < duomenys_gijoms[i].n; j++)
		{
			for (int k = 0; k < MAX_ARRAY_SIZE; k++)
			{
				if (strcmp(result_array[k].text_var, "") == 0)
				{
					strcpy(result_array[k].text_var, duomenys_gijoms[i].thread_struct_array[j].text_var);
					break;
				}
				else if (strcmp(result_array[k].text_var, duomenys_gijoms[i].thread_struct_array[j].text_var) == 0)
				{
					break;
				}
			}
		}
	}
	addWithCuda(result_array, duomenys_gijoms);
	for (int i = 0; i < MAX_ARRAY_SIZE; i++)
	{
		printf("%10s %10d %10lf\n", result_array[i].text_var, result_array[i].int_var, result_array[i].double_var);
	}
	printf("Press any key to continue...");
	fgetchar();
	hipDeviceReset();
	return 0;
}

void addWithCuda(Data *c, ThreadData *a)
{
	//thrust::host_vector<ThreadData> H(MAX_ARRAY_SIZE);
	//thrust::host_vector<Data> CH(MAX_ARRAY_SIZE);
	//for (int i = 0; i < MAX_ARRAY_SIZE; i++)	{H[i] = a[i];}
	//for (int i = 0; i < MAX_ARRAY_SIZE; i++)		{CH[i] = c[i];}
	//thrust::device_vector<Data> D = H;
	//thrust::device_vector<ThreadData> CD;
	thrust::host_vector<Data> H1(5);
	for (int i = 0; i < 5; i++)	{H1[i] = a[0].thread_struct_array[i];}
	thrust::host_vector<Data> H2(5);
	for (int i = 0; i < 5; i++)	{H2[i] = a[1].thread_struct_array[i];}
	thrust::host_vector<Data> H3(5);
	for (int i = 0; i < 5; i++)	{H3[i] = a[2].thread_struct_array[i];}
	thrust::host_vector<Data> H4(5);
	for (int i = 0; i < 5; i++)	{H4[i] = a[3].thread_struct_array[i];}
	thrust::host_vector<Data> H5(5);
	for (int i = 0; i < 5; i++)	{H5[i] = a[4].thread_struct_array[i];}

	printf("%10s\n", H1.size());
	/*thrust::device_vector<Data> D1 = H1;
	thrust::device_vector<Data> D2 = H2;
	thrust::device_vector<Data> D3 = H3;
	thrust::device_vector<Data> D4 = H4;
	thrust::device_vector<Data> D5 = H5;
	thrust::device_vector<Data> D(MAX_ARRAY_SIZE);
	for (int i = 0; i < MAX_ARRAY_SIZE; i++)
	{
		D[i] = Plius(D1[i], D2[i], D3[i], D4[i], D5[i]);
	}
	thrust::host_vector<Data> H = D;*/
}

Data Plius(Data D1, Data D2, Data D3, Data D4, Data D5)
{
	Data result;
	strcpy(result.text_var, D1.text_var);
	strcpy(result.text_var, D2.text_var);
	strcpy(result.text_var, D3.text_var);
	strcpy(result.text_var, D4.text_var);
	strcpy(result.text_var, D5.text_var);
	result.int_var = D1.int_var + D2.int_var + D3.int_var + D4.int_var + D5.int_var;
	result.double_var = D1.double_var + D2.double_var + D3.double_var + D4.double_var + D5.double_var;
	return result;
}