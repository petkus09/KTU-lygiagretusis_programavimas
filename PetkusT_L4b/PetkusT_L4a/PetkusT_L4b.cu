//IFF-1 Tautvydas Petkus
// L4b - Thrust
//Failo dydis - 50 eiluciu
//Dabartiniai nustatymai: giju sk: 5, maximalus masyvo dydis - 5, didziausias char buferio dydis - 70
//Kiek  iteracij� i� eil�s padaro vienas procesas? viena pilnai
//Kokia tvarka vykdomi procesai? tokia, kokia startuoja
//

#include "hip/hip_runtime.h"
//#include ""

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>	
#include <fstream>	
#include <sstream>	
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
//#include <thrust/sort.h>
//#include <thrust/sequence.h> 

using namespace std;

const int MAX_STRING_LEN = 50;
const int MAX_THREADS = 5;
const int MAX_FILE_ROW = 50;
const int MAX_ARRAY_SIZE = 5;

struct Data
{
	char text_var[MAX_STRING_LEN];
	int int_var;
	double double_var;
};

struct ThreadData{
	struct Data thread_struct_array[MAX_ARRAY_SIZE];
	int n;
};

Data Plus(Data D1, Data D2, Data D3, Data D4, Data D5);
void addWithCuda(ThreadData *a);

int main()
{
	//pradiniu duomenu kintamieji
	int array_size[MAX_THREADS] = {5, 5, 5, 5, 5};
	ifstream in("PetkusT.txt");
	struct Data duomenys[MAX_FILE_ROW];
	struct ThreadData duomenys_gijoms[MAX_THREADS];
	//Skaitomas duomenu failas
	for(string line; getline(in, line);){	
		for (int i = 0; i < MAX_FILE_ROW; i++)
		{
			getline(in, line);
			if (line != ""){
				stringstream ss;
				ss << line;
				Data eilute;
				char string_var[16];
				ss >> string_var >> eilute.int_var >> eilute.double_var;

				strcpy(eilute.text_var, string_var);
				duomenys[i] = eilute;
			}
		}
	}
	//pradiniu duomenu atspausdinimas
	int d = 0;
	int ii = 0;
	int j = 0;
	printf("********************************************************************\n");
	printf("***Pradiniai duomenys***\n");
	for (ii = 0; ii < MAX_THREADS; ii++){
		printf("***Gija nr. %d***\n", ii + 1);
		printf("%10s %10s %10s %10s\n", "Eil.Nr.", "String", "int", "double");
		struct Data D_gija[MAX_ARRAY_SIZE];
		for (j = 0; j < array_size[ii]; j++){
			D_gija[j] = duomenys[d];
			printf("%10d %10s %10d %10lf\n", j + 1, D_gija[j].text_var, D_gija[j].int_var, D_gija[j].double_var);
			d++;

		}
		for (j = 0; j < array_size[ii]; j++){
			duomenys_gijoms[ii].thread_struct_array[j] = D_gija[j];
			duomenys_gijoms[ii].n = array_size[ii];
		}
	}
	printf("\n**********************\n");
	//Lygiagrecioji dalis

	printf("***Lygiagrecioji programos dalis***\n");
	printf("%50s %10s %10s\n", "String", "int", "double");
	addWithCuda(duomenys_gijoms);
	printf("Press any key to continue...");
	fgetchar();
	return 0;
}

void addWithCuda(ThreadData *a)
{

	thrust::host_vector<Data> H1(5);
	H1[0] = a[0].thread_struct_array[0];
	H1[1] = a[0].thread_struct_array[1];
	H1[2] = a[0].thread_struct_array[2];
	H1[3] = a[0].thread_struct_array[3];
	H1[4] = a[0].thread_struct_array[4];
	thrust::device_vector<Data> D1 = H1;

	thrust::host_vector<Data> H2(5);
	H2[0] = a[1].thread_struct_array[0];
	H2[1] = a[1].thread_struct_array[1];
	H2[2] = a[1].thread_struct_array[2];
	H2[3] = a[1].thread_struct_array[3];
	H2[4] = a[1].thread_struct_array[4];
	thrust::device_vector<Data> D2 = H2;

	thrust::host_vector<Data> H3(5);
	H3[0] = a[2].thread_struct_array[0];
	H3[1] = a[2].thread_struct_array[1];
	H3[2] = a[2].thread_struct_array[2];
	H3[3] = a[2].thread_struct_array[3];
	H3[4] = a[2].thread_struct_array[4];
	thrust::device_vector<Data> D3 = H3;

	thrust::host_vector<Data> H4(5);
	H4[0] = a[3].thread_struct_array[0];
	H4[1] = a[3].thread_struct_array[1];
	H4[2] = a[3].thread_struct_array[2];
	H4[3] = a[3].thread_struct_array[3];
	H4[4] = a[3].thread_struct_array[4];
	thrust::device_vector<Data> D4 = H4;

	thrust::host_vector<Data> H5(5);
	H5[0] = a[4].thread_struct_array[0];
	H5[1] = a[4].thread_struct_array[1];
	H5[2] = a[4].thread_struct_array[2];
	H5[3] = a[4].thread_struct_array[3];
	H5[4] = a[4].thread_struct_array[4];
	thrust::device_vector<Data> D5 = H5;

	thrust::device_vector<Data> D(MAX_ARRAY_SIZE);
	for (int i = 0; i < 5; i++)
	{
		D[i] = Plus(D1[i], D2[i], D3[i], D4[i], D5[i]);
	}
	thrust::host_vector<Data> H = D;
	for (int i = 0; i < 5; i++)
	{
		printf("%50s %10d %10lf\n", H[i].text_var, H[i].int_var, H[i].double_var);
	}
}

Data Plus(Data D1, Data D2, Data D3, Data D4, Data D5)
{
	Data result;
	/*for (int i = 0; i < 10; i++)
	{
		result.text_var[i] = D1.text_var[i];
	}
	for (int i = 0; i < 10; i++)
	{
		result.text_var[i+10] = D2.text_var[i];
	}
	for (int i = 0; i < 10; i++)
	{
		result.text_var[i+20] = D3.text_var[i];
	}	for (int i = 0; i < 10; i++)
	{
		result.text_var[i+30] = D4.text_var[i];
	}
	for (int i = 0; i < 10; i++)
	{
		result.text_var[i+40] = D5.text_var[i];
	}*/
	strcpy(result.text_var, D1.text_var);
	strcat(result.text_var, D2.text_var);
	strcat(result.text_var, D3.text_var);
	strcat(result.text_var, D4.text_var);
	strcat(result.text_var, D5.text_var);
	/*strcpy(result.text_var, D2.text_var);
	strcpy(result.text_var, D3.text_var);
	strcpy(result.text_var, D4.text_var);
	strcpy(result.text_var, D5.text_var);*/
	result.int_var = D1.int_var + D2.int_var + D3.int_var + D4.int_var + D5.int_var;
	result.double_var = D1.double_var + D2.double_var + D3.double_var + D4.double_var + D5.double_var;
	return result;
}