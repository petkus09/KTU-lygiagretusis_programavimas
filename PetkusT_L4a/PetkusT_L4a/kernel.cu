#include "hip/hip_runtime.h"
//IFF-1 Tautvydas Petkus
// L4a - CUDA
//Failo dydis - 50 eiluciu
//Dabartiniai nustatymai: giju sk: 8, maximalus masyvo dydis - 10, didziausias char buferio dydis - 80
//Kiek  iteracij� i� eil�s padaro vienas procesas? viena pilnai
//Kokia tvarka vykdomi procesai? tokia, kokia startuoja
//

#include "hip/hip_runtime.h"
#include ""

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>	
#include <fstream>	
#include <sstream>	
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

using namespace std;

const int MAX_STRING_LEN = 16;
const int MAX_THREADS = 8;
const int MAX_FILE_ROW = 50;
const int MAX_ARRAY_SIZE = 10;

struct Data
{
	char text_var[MAX_STRING_LEN];
	int int_var;
	double double_var;
};

struct ThreadData{
	struct Data thread_struct_array[MAX_ARRAY_SIZE];
	int n;
};

hipError_t addWithCuda(Data *c, ThreadData *a);

__global__ void addKernel(Data *c, ThreadData *a)
{
	int i = threadIdx.x;
	for (int j = 0; j < a[i].n; j++)
	{
		for (int k = 0; k < MAX_ARRAY_SIZE; k++)
		{
			bool check = true;
			for (int cc = 0; cc < MAX_STRING_LEN; cc++)
			{
				if (c[k].text_var[cc] != a[i].thread_struct_array[j].text_var[cc])
				{
					check = false;
				}
			}
			if (check)
			{
				c[k].int_var += a[i].thread_struct_array[j].int_var;
				c[k].double_var += a[i].thread_struct_array[j].double_var;
			}
		}
	}
}

int main()
{
	//pradiniu duomenu kintamieji
	int array_size[MAX_THREADS] = {5, 7, 6, 9, 4, 8, 4, 6};
	ifstream in("PetkusT.txt");
	struct Data duomenys[MAX_FILE_ROW];
	struct ThreadData duomenys_gijoms[MAX_THREADS];
	//Skaitomas duomenu failas
	for(string line; getline(in, line);){	
		for (int i = 0; i < MAX_FILE_ROW; i++)
		{
			getline(in, line);
			if (line != ""){
				stringstream ss;
				ss << line;
				Data eilute;
				char string_var[16];
				ss >> string_var >> eilute.int_var >> eilute.double_var;

				strcpy(eilute.text_var, string_var);
				duomenys[i] = eilute;
			}
		}
	}
	//pradiniu duomenu atspausdinimas
	int d = 0;
	int ii = 0;
	int j = 0;
	printf("********************************************************************\n");
	printf("***Pradiniai duomenys***\n");
	for (ii = 0; ii < MAX_THREADS; ii++){
		printf("***Gija nr. %d***\n", ii + 1);
		printf("%10s %10s %10s %10s\n", "Eil.Nr.", "String", "int", "double");
		struct Data D_gija[MAX_ARRAY_SIZE];
		for (j = 0; j < array_size[ii]; j++){
			D_gija[j] = duomenys[d];
			printf("%10d %10s %10d %10lf\n", j + 1, D_gija[j].text_var, D_gija[j].int_var, D_gija[j].double_var);
			d++;

		}
		for (j = 0; j < array_size[ii]; j++){
			duomenys_gijoms[ii].thread_struct_array[j] = D_gija[j];
			duomenys_gijoms[ii].n = array_size[ii];
		}
	}
	printf("\n**********************\n");
	//Lygiagrecioji dalis

	printf("***Lygiagrecioji programos dalis***\n");
	printf("%10s %10s %10s\n", "String", "int", "double");
	struct Data result_array[MAX_ARRAY_SIZE];
	for (int i = 0; i < MAX_ARRAY_SIZE; i++)
	{
		Data duom;
		strcpy(duom.text_var, "");
		duom.int_var = 0;
		duom.double_var = 0.0;
		result_array[i] = duom;
	}
	for (int i = 0; i < MAX_THREADS; i++)
	{
		for (int j = 0; j < duomenys_gijoms[i].n; j++)
		{
			for (int k = 0; k < MAX_ARRAY_SIZE; k++)
			{
				if (strcmp(result_array[k].text_var, "") == 0)
				{
					strcpy(result_array[k].text_var, duomenys_gijoms[i].thread_struct_array[j].text_var);
					break;
				}
				else if (strcmp(result_array[k].text_var, duomenys_gijoms[i].thread_struct_array[j].text_var) == 0)
				{
					break;
				}
			}
		}
	}
	
	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(result_array, duomenys_gijoms);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		fgetchar();
		return 1;
	}
	for (int i = 0; i < MAX_ARRAY_SIZE; i++)
	{
		printf("%10s %10d %10lf\n", result_array[i].text_var, result_array[i].int_var, result_array[i].double_var);
		//printf(" %10d %10lf\n", result_array[i].int_var, result_array[i].double_var);
	}
	printf("Press any key to continue...");
	fgetchar();
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(Data *c, ThreadData *a)
{
	ThreadData *dev_a = 0;
	Data *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)
	cudaStatus = hipMalloc((void**)&dev_c, MAX_ARRAY_SIZE *  sizeof(Data));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, MAX_THREADS * sizeof(ThreadData));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	//cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	//if (cudaStatus != hipSuccess) {
	//    fprintf(stderr, "hipMalloc failed!");
	//    goto Error;
	//}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, MAX_THREADS * sizeof(ThreadData), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_c, c, MAX_ARRAY_SIZE *  sizeof(Data), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	//cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	//if (cudaStatus != hipSuccess) {
	//    fprintf(stderr, "hipMemcpy failed!");
	//    goto Error;
	//}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel<<<1, MAX_THREADS - 1>>>(dev_c, dev_a);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c,  MAX_ARRAY_SIZE * sizeof(Data), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);

	return cudaStatus;
}
