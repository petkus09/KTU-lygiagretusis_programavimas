
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <istream>
#include <iostream>
#include <fstream>
#include <sstream>

using namespace std;

const int MAX_STRING_LEN = 80;
const int MAX_THREADS = 8;
const int MAX_FILE_ROW = 50;
const int MAX_ARRAY_SIZE = 10;

struct Data
{
   char text_var[MAX_STRING_LEN];
   int int_var;
   double double_var;
};

struct ThreadData{
    struct Data thread_struct_array[MAX_ARRAY_SIZE];
};

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
	//Nustatoma, kiek kiekviena gija tures masyvo elementu
    int array_size[MAX_THREADS] = {5, 7, 6, 9, 4, 8, 4, 6};
    struct Data duomenys[MAX_FILE_ROW];

    //Nuskaitomo failo paruosimas
    //char * line = NULL;
    //size_t len = 0;
    //ssize_t read;
    //FILE *ifp;
    //char *mode = "r";
    //ifp = fopen("PetkusT.txt", mode);
    //if (ifp == NULL) {
    //    fprintf(stderr, "Can't open input file in.list!\n");
     //   exit(1);
    //}
	ifstream fin("PetkusT.txt");
    char ch;
    while (fin.get(ch)){
	//string ch;
	//while( getline(fin, ch) ) {  
        printf("%s\n", ch);
	}
    fin.close();


    /*char A1[MAX_STRING_LEN];
    int A2;
    double A3;
    int n;
    int i = 0;
    read = getline(&line, &len, ifp);
    //Duomenu nuskaitymas i viena bendra masyva
    while ((read = getline(&line, &len, ifp)) != -1 && i < MAX_FILE_ROW) {
           n = sscanf(line,"%s %d %lf",A1,&A2,&A3);
           struct Data kintamasis = {.int_var = A2, .double_var = A3};
           strncpy(kintamasis.text_var, A1, MAX_STRING_LEN);
           duomenys[i] = kintamasis;
           i = i + 1;
       }*/

    //Duomenu priskyrimas giju masyvams. Pradiniu duomenu isvedimas
    /*int d = 0;
    int ii = 0;
    int j = 0;
    printf("********************************************************************\n");
    printf("***Pradiniai duomenys***\n");
    struct ThreadData duomenys_gijoms[MAX_THREADS];
    for (ii = 0; ii < MAX_THREADS; ii++){
        printf("***Gija nr. %d***\n", ii + 1);
        printf("%10s %10s %10s %10s\n", "Eil.Nr.", "String", "int", "double");
        struct Data D_gija[array_size[ii]];
        for (j = 0; j < array_size[ii]; j++){
            D_gija[j] = duomenys[d];
            printf("%10d %10s %10d %10lf\n", j + 1, D_gija[j].text_var, D_gija[j].int_var, D_gija[j].double_var);
            d++;

        }
        for (j = 0; j < array_size[ii]; j++){
            duomenys_gijoms[ii].thread_struct_array[j] = D_gija[j];
        }
    }
    printf("\n**********************\n");*/

    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
	
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
	
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
