//IFF-1 Tautvydas Petkus
// L1d - CUDA
//Failo dydis - 50 eiluciu
//Dabartiniai nustatymai: giju sk: 8, maximalus masyvo dydis - 10, didziausias char buferio dydis - 80
//Kiek  iteracij� i� eil�s padaro vienas procesas? viena pilnai
//Kokia tvarka vykdomi procesai? tokia, kokia startuoja
//

#include "hip/hip_runtime.h"




#include <stdio.h>
#include <iostream>	
#include <fstream>	
#include <sstream>	

using namespace std;

const int MAX_STRING_LEN = 16;
const int MAX_THREADS = 8;
const int MAX_FILE_ROW = 50;
const int MAX_ARRAY_SIZE = 10;


#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200)
  # error printf is only supported on devices of compute capability 2.0 and higher, please compile with -arch=sm_20 or higher    
#endif


//duomenis saugantis struct formatas
struct Data
{
   char text_var[MAX_STRING_LEN];
   int int_var;
   double double_var;
};

struct ThreadData{
    struct Data thread_struct_array[MAX_ARRAY_SIZE];
};

//gijos spausdinimo funkcija
__global__ void Thread_Print(ThreadData *duomenys_gijoms, int array_size[MAX_ARRAY_SIZE]){
	int gijosNr = threadIdx.x;
	int j = 0;
        for (j = 0; j < array_size[gijosNr]; j++){
            printf("%10s%d %10d %10s %10d %10lf\n","Procesas", gijosNr + 1, j + 1, duomenys_gijoms[gijosNr].thread_struct_array[j].text_var, duomenys_gijoms[gijosNr].thread_struct_array[j].int_var, duomenys_gijoms[gijosNr].thread_struct_array[j].double_var);
            int ii = 0;
            //funkcija, reikalinga pristabdyti giju veikima ir pastebeti maisos rezultatus
            //for (ii = 0; ii < 1000; ii++){
            //    double bandomasis = ii * ii * ii * ii * ii * ii * ii * ii;
            //}
        }
        printf("***Gija nr. %d baige darba \n", gijosNr + 1);
	
}
// Helper function for using CUDA to add vectors in parallel.
__global__ void addKernel(int *c, const int *a, const int *b, int *ind)
{
    int i = threadIdx.x;
	ind[i] = i;
    c[i] = a[i] + b[i];

	//cuPrintf("V alue: %d\n", i);

}

hipError_t runAll(ThreadData *a, int *size){ //lygiagrecioji dalis
	ThreadData *dev_c = 0;
	int *i =0;
	hipError_t cudaStatus;

	//perkeliame duomenis i� CPU � vaizdo plok�t�
	cudaStatus = hipMalloc((void**)&dev_c, MAX_THREADS * sizeof(ThreadData));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(dev_c, a, MAX_THREADS * sizeof(ThreadData), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&i, MAX_ARRAY_SIZE * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(i, size, MAX_ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	//paleid�iame gijas
    Thread_Print<<<1, MAX_THREADS - 1>>>(dev_c, i);

	Error:
		hipFree(dev_c);
		hipFree(i);
	return cudaStatus;
}

int main()
{
	//pradiniu duomenu kintamieji
	int array_size[MAX_THREADS] = {5, 7, 6, 9, 4, 8, 4, 6};
	ifstream in("PetkusT.txt");
	struct Data duomenys[MAX_FILE_ROW];
	struct ThreadData duomenys_gijoms[MAX_THREADS];
	//Skaitomas duomenu failas
	for(string line; getline(in, line);){	
		for (int i = 0; i < MAX_FILE_ROW; i++)
		{
			getline(in, line);
			if (line != ""){
				stringstream ss;
				ss << line;
				Data eilute;
				char string_var[16];
				ss >> string_var >> eilute.int_var >> eilute.double_var;
				
				strcpy(eilute.text_var, string_var);
				duomenys[i] = eilute;
			}
		}
	}
	//pradiniu duomenu atspausdinimas
	int d = 0;
    int ii = 0;
    int j = 0;
    printf("********************************************************************\n");
    printf("***Pradiniai duomenys***\n");
    for (ii = 0; ii < MAX_THREADS; ii++){
        printf("***Gija nr. %d***\n", ii + 1);
        printf("%10s %10s %10s %10s\n", "Eil.Nr.", "String", "int", "double");
        struct Data D_gija[MAX_ARRAY_SIZE];
        for (j = 0; j < array_size[ii]; j++){
            D_gija[j] = duomenys[d];
            printf("%10d %10s %10d %10lf\n", j + 1, D_gija[j].text_var, D_gija[j].int_var, D_gija[j].double_var);
            d++;

        }
        for (j = 0; j < array_size[ii]; j++){
            duomenys_gijoms[ii].thread_struct_array[j] = D_gija[j];
        }
    }
    printf("\n**********************\n");
	//Lygiagrecioji dalis

	printf("***Lygiagrecioji programos dalis***\n");
    printf("%10s %10s %10s %10s %10s\n", "Gijos nr.", "Eil.Nr.", "String", "int", "double");
	runAll(duomenys_gijoms, array_size); //Vyksta CUDA spausdinimas
	return 0;
}

